#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : GScuda.cu
 Author      : caleb
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdint.h>

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err);

__global__ void MWKernel(uint8_t n, uint8_t* male_prefs, uint8_t* female_prefs, uint8_t* output) {
    __shared__ current_match[n]; 
	//find a female to propose to
	male_prefs[thread]
}

/**
 * Host function that copies the data and launches GS on the CPU
 *
 */
void  MWcuda(uint8_t n, uint8_t* male_prefs, uint8_t* female_prefs, uint8_t* output)
{
	int *d_male_prefs, *d_female_prefs,*d_fast_female;
	char *d_is_engaged;
	int *d_next_female;
	int* d_output;

	size_t prefs_size = sizeof(int)*n*n;
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_male_prefs, prefs_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_female_prefs, prefs_size));
	//CUDA_CHECK_RETURN(hipMalloc((void **)&d_fast_female, prefs_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_output, sizeof(int)*n));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_is_engaged, sizeof(char)*n));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_next_female, sizeof(char)*n));
	CUDA_CHECK_RETURN(hipMemcpy(d_male_prefs, male_prefs, prefs_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_female_prefs, female_prefs, prefs_size, hipMemcpyHostToDevice));


//	FFKernel<<<n,n>>>  (n,d_female_prefs,d_fast_female);

	//static const int BLOCK_SIZE = 256;
	//const int blockCount = (size+BLOCK_SIZE-1)/BLOCK_SIZE;
	// make fast_female


	MWKernel<<<1,n>>> (n, d_male_prefs,d_female_prefs, d_output);

	CUDA_CHECK_RETURN(hipMemcpy(output, d_output, sizeof(int)*n, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_male_prefs));
	CUDA_CHECK_RETURN(hipFree(d_female_prefs));
	CUDA_CHECK_RETURN(hipFree(d_fast_female));
	CUDA_CHECK_RETURN(hipFree(d_output));
}
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}




