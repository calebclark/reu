#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : GScuda.cu
 Author      : caleb
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdint.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>
#include <hipcub/hipcub.hpp>
using namespace hipcub;

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err);

__global__ void piiKernel(uint8_t n, uint8_t* male_prefs, uint8_t* female_prefs, uint8_t* output) {
    int idx = threadIdx.x;
    extern __shared__ int current_match[]; 
    // initialize random number generator
    hiprandState state;
    hiprand_init(idx,0,0,&state);

}


#ifdef DEBUG
    if (idx == 0) {
        bool* contains = new bool[n];
        bool bad = false;
        for (int i = 0; i < n; i++)
            contains[i] = 0;
        for (int i = 0; i < n;i++) {
            if (current_match[i] > n || contains[current_match[i]])
                bad = true;
            contains[current_match[i]] = true;
            printf("%d ",current_match[i]);
        }
        if (!bad)
            printf("\npermutation is good\n");
        else
            printf("\npermutation is bad\n");
        delete[] contains;
    }
#endif




}

/**
 * Host function that copies the data and launches GS on the CPU
 *
 */
void  pii(uint8_t n, uint8_t* male_prefs, uint8_t* female_prefs, uint8_t* output)
{
	uint8_t *d_male_prefs, *d_female_prefs;
	uint8_t* d_output;

	size_t prefs_size = sizeof(uint8_t)*n*n;
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_male_prefs, prefs_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_female_prefs, prefs_size));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_output, sizeof(uint8_t)*n));
	CUDA_CHECK_RETURN(hipMemcpy(d_male_prefs, male_prefs, prefs_size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_female_prefs, female_prefs, prefs_size, hipMemcpyHostToDevice));


    struct timespec start, end;
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &start);	
    piiKernel<<<1,n,n*sizeof(int)>>> (n, d_male_prefs,d_female_prefs, d_output);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);	
    long long unsigned int diff = (1000000000L) * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;

    printf("kernel time %llu\n",diff);


	CUDA_CHECK_RETURN(hipMemcpy(output, d_output, sizeof(uint8_t)*n, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_male_prefs));
	CUDA_CHECK_RETURN(hipFree(d_female_prefs));
	CUDA_CHECK_RETURN(hipFree(d_output));
}
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}




