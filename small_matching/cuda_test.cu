#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : GScuda.cu
 Author      : caleb
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdint.h>
#include <limits.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <time.h>
#include <assert.h>
#include <hipcub/hipcub.hpp>
using namespace hipcub;
// I ASSUME THROUGHOUT THAT sizeof(int) = 4
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err);

__global__ void emptyKernel() {
}

/**
 * Host function that copies the data and launches GS on the CPU
 *
 */
void  empty_kernel()
{

    struct timespec start, end;
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &start);	
    emptyKernel<<<1,1>>> ();
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end);	
    long long unsigned int diff = (1000000000L) * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec;
    printf("kernel time %llu\n",diff);
}
/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}




